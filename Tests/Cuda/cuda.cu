#include "hip/hip_runtime.h"
/**
 * @todo 1. Superzellen manager verbessern. Oder besser Richtig implementieren.
 * 
 */
#include "Tests/Cuda/cuda.hpp"
#include "PIC/SupercellManager.hpp"
#include "PIC/SupercellContainer.hpp"
#include "PIC/SupercellContainerManager.hpp"
#undef _OPENMP
#include "Iterator/RuntimeTuple.hpp"
#include "View.hpp"
/***************************************************************
 * first Test case: Add a one to all particles first attribute
 * ******************************************************************/
template<typename T=void>
__global__
void 
FrameInSuperCell(Supercell *supercell, const int nbParticleInLastFrame)
{  
   
    typedef typename Supercell::FrameType Frame;
    const int jumpsizeParticle = 256;
    const int offsetParticle = threadIdx.x;
    const int nbElementsParticle = nbParticleInLastFrame;
    typedef hzdr::runtime::TupleFull RuntimeTuple;
    
    const RuntimeTuple runtimeVarParticle(offsetParticle, nbElementsParticle, jumpsizeParticle);
    
    
    const int jumpsizeFrame = 1;
    const int offsetFrame = 0;
    const int nbElementsFrame = 0;
    const RuntimeTuple runtimeFrame(offsetFrame, nbElementsFrame, jumpsizeFrame);
    
    typedef hzdr::View<Frame, hzdr::Direction::Forward,  hzdr::Collectivity::None,RuntimeTuple> ParticleInFrame;
    
    hzdr::View<Supercell, hzdr::Direction::Forward,  hzdr::Collectivity::CudaIndexable, RuntimeTuple, ParticleInFrame> view(supercell, runtimeFrame, ParticleInFrame(nullptr, runtimeVarParticle)); 
    
     auto it=view.begin();

     for(auto it=view.begin(); it!=view.end(); ++it)
     {
         if(*it)
         {
             (**it).data[0] += 1;
        }
     }
}

/**
 * @brief 
 */

void
callSupercellAddOne(Supercell** supercell, int Frames, int nbParticleInFrame)
{
    SupercellHandle<Supercell> supercellHandler(Frames, nbParticleInFrame);


    FrameInSuperCell<<<1, 256>>>(supercellHandler.supercellGPU, nbParticleInFrame);
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    supercellHandler.copyDeviceToHost();
    *supercell = new Supercell(*(supercellHandler.supercellCPU));

}


/***************************************************************
 * second Test case: Add all Particles of the Supercell together
 * *************************************************************/
template<typename T=void>
__global__
void 
addAllParticlesInSupercell(Supercell *supercell, const int nbSupercells)
{
    // define all needed types 
    typedef hzdr::SupercellContainer<Supercell> SupercellContainer;
    typedef hzdr::runtime::TupleFull RuntimeTuple;
    typedef typename Supercell::FrameType Frame;
    typedef hzdr::View<SupercellContainer, 
                       hzdr::Direction::Forward, 
                       hzdr::Collectivity::None, 
                       RuntimeTuple> ViewSupercellContainer;
    typedef hzdr::View<Frame, 
                       hzdr::Direction::Forward,  
                       hzdr::Collectivity::None,
                       RuntimeTuple> ParticleInFrame;
    typedef  hzdr::View<Supercell,
                        hzdr::Direction::Forward,  
                        hzdr::Collectivity::CudaIndexable, 
                        RuntimeTuple, 
                        ParticleInFrame> FrameInSupercellView;
    // define shared variables
    __shared__ int32_t mem[256];
    __shared__ int32_t result;
    
    // create the iteratable container.
    SupercellContainer supercellContainer(supercell, nbSupercells);  

    // create the first second: over all supercells
    const int jumpsizeSupercells = 1;
    const int offsetSupercells = 0;
    const RuntimeTuple runSupercell(offsetSupercells, nbSupercells, jumpsizeSupercells);
    ViewSupercellContainer viewSupercellContainer(supercellContainer, runSupercell);
    
    
    
    for(auto itSupercell=viewSupercellContainer.begin();
        itSupercell != viewSupercellContainer.end();
        ++itSupercell)
    {
        if(*itSupercell)// and *(itSupercell+1))
        {
            
            // create the second view: over all frames within the supercell
            const int jumpsizeParticle = 256;
            const int offsetParticle = threadIdx.x;
            const int nbElementsParticle1 = (**itSupercell).nbParticlesInLastFrame;
            const int nbElementsParticle2 = (**(itSupercell+1)).nbParticlesInLastFrame;
            
            const RuntimeTuple runtimeFrame(0, 0, 1);
            const RuntimeTuple runtimeVarParticle(offsetParticle, nbElementsParticle1, jumpsizeParticle);
            FrameInSupercellView view(**itSupercell, runtimeFrame, ParticleInFrame(nullptr, runtimeVarParticle)); 
        
            const RuntimeTuple runtimeVarParticle2(offsetParticle, nbElementsParticle1, jumpsizeParticle);
            FrameInSupercellView view2(**(itSupercell+1), runtimeFrame, ParticleInFrame(nullptr, runtimeVarParticle2));
            
            result = 0;
            for(auto it2=view2.begin(); it2!=view2.end(); ++it2)
            {
                if(*it2)
                {
                   //  (**it2).data[1] = 1; 
                    mem[threadIdx.x] = (**it2).data[0];
                    __syncthreads();
                    atomicAdd(&result, mem[threadIdx.x]);
                }
            }
                        

            for(auto it=view.begin(); it!=view.end(); ++it)
            {
                if(*it)
                {
                    (**it).data[0] = result;
                }
            }    
            
           
        }
        

        
        
        

        
    }
    

}

void callSupercellSquareAdd(Supercell*** superCellContainer, int nbSupercells, std::vector<int> nbFramesSupercell, std::vector<int> nbParticlesInFrame)
{
    SupercellContainerManager<Supercell> supercellHandler(nbSupercells, nbFramesSupercell, nbParticlesInFrame);
    addAllParticlesInSupercell<<<1, 256>>>(supercellHandler.supercellGPU, nbSupercells);
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipPeekAtLastError() );
    supercellHandler.copyDeviceToHost();
    
    *superCellContainer = new Supercell*[nbSupercells];
    for(int i=0; i<nbSupercells; ++i)
    {
        (*superCellContainer)[i] = &(supercellHandler.supercellCPU[i]);
    }
    
}
